#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdio>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "kernels.h"
#include "time_measure.h"
#include "common.h"

void vypsat_mat(int nx, int ny, unsigned int* matice, unsigned int* prava_strana)
{
	printf("\n");
	for(int y=0;y<min(ny,12);y++)
	{
		int x;
		for(x=0;x<min(nx,8);x++)
		{
			unsigned int a=matice[x+y*nx];
			printf("%6u\t", a);
		}
		if(x<nx-1)
		{
			printf("...");
		}
		printf("| ");
		if(prava_strana!=NULL)
		{
			printf("%u", prava_strana[y]);
		}
		printf("\n");
	}
}
int save_matrix(int N, unsigned int* matice, unsigned int* prava_strana, FILE* f)
{
	/*fstream file;
	file.open(filename, fstream::out);
	if(!file.is_open()) return 1;*/
	
	 if (f==NULL) return 1;
	//file << N << endl;
	fprintf(f, "%d\n", N);
	
	for(int y=0;y<N;y++)
	{
		int x;
		for(x=0;x<N;x++)
		{
			//file << matice[get_index(x, y, N)] << "\t";
			fprintf(f, "%8u\t", matice[x+y*N]);
		}
		if(prava_strana!=NULL)
		{
			//file << "| " << prava_strana[y];
			fprintf(f, "| %u", prava_strana[y]);
		}
		//file << endl;
		fprintf(f, "\n");
	}
	return 0;
}


__device__ int cuda_get_index(int X, int Y, int N)	// SLOUPEC, RADEK
{
	return Y*N+X;
}
__device__ unsigned int cuda_compute_inverse_eukleides(unsigned int cislo, unsigned int modul)
{
	unsigned int a, b, a1, b1, q, r;
	a = cislo;
	b = modul;
	a1 = 0;
	b1 = 1;
	int plus = 1;

	while( b!=0 )
	{
		q = a / b;
		r = a % b;
		a = b;
		b = r;
		r = a1;
		a1 = b1 + r*q;
		b1 = r;
		plus=-plus;
	}
	if( a==1 )
	{
		if( 0<plus )
		{
			return (unsigned int)b1;
		}else
		{
			return (unsigned int)(modul-b1);
		}
	}
	return (unsigned int)0;
}
__device__ unsigned int cuda_multiply_add_modulo(unsigned int modul, unsigned int a, unsigned int b, unsigned int c)
// \STATE := (a * b + c) % modul
{
	// integer aritmetika
	unsigned long long pom = a;
	pom *= b;
	pom += c;
	pom %= modul;
	return (unsigned int)pom;

	// float aritmetika
	double p1;
	p1 = ((double)a) * ((double)b) + ((double)c);
	// __fma_rd(a, b, c) k vypoctu (a * b + c)
	//p1 = __fma_rd( (double)a, (double)b, (double)c );

	double q = floor( p1/modul );
	double p2 = q * modul;
	return ((unsigned int)(p1 - p2));
}

// elementarni uprava bez deleni
__device__ unsigned int cuda_elem_uprava_bez_deleni(unsigned int modul, unsigned int a_xy, unsigned int a_pp, unsigned int a_xp, unsigned int a_py)
// \STATE $a_{xy} := a_{xy} \cdot a_pp - a_xp \cdot a_py$
{
	unsigned long long m1;
	unsigned long long pom;

	pom = a_xy;
	m1 = a_xp;
	
	pom *= a_pp;
	m1 *= a_py;
	if(pom >= m1)
	{
		pom -= m1;
		pom %= modul;
	}else
	{
		m1 -= pom;
		m1 %= modul;
		pom = modul-m1;
	}
	return ((unsigned int)pom);
}
// elementarni uprava s delenim
__device__ unsigned int cuda_elem_uprava_s_delenim(unsigned int modul, unsigned int a_xy, unsigned int a_xp, unsigned int a_py)
// \STATE $a_{xy} := a_{xy} - a_xp \cdot a_py$
{
	unsigned long long m1;
	unsigned long long pom;

	pom = a_xy;
	m1 = a_xp;
	
	m1 *= a_py;
	if(pom >= m1)
	{
		pom -= m1;
		pom %= modul;
	}else
	{
		m1 -= pom;
		m1 %= modul;
		pom = modul-m1;
	}
	return ((unsigned int)pom);
}
// elementarni uprava bez deleni
__device__ unsigned int cuda_elem_uprava_bez_deleni1(unsigned int modul, unsigned int a_xy, unsigned int a_pp, unsigned int a_xp, unsigned int a_py)
// \STATE $a_{xy} := a_{xy} \cdot a_pp - a_xp \cdot a_py$
{
	double p1 = (((double)a_xy) * ((double)a_pp)) - (((double)a_xp) * ((double)a_py));
	double q = floor(p1/(double)modul);
	double p2 = q*modul;
	
	return ((unsigned int)(p1-p2));
}
// elementarni uprava s delenim
__device__ unsigned int cuda_elem_uprava_s_delenim1(unsigned int modul, unsigned int a_xy, unsigned int a_xp, unsigned int a_py)
// \STATE $a_{xy} := a_xy - a_xp \cdot a_py$
{
	double p1 = ((double)a_xy) - (((double)a_xp) * ((double)a_py));
	double q = floor(p1/(double)modul);
	double p2 = q*modul;
	
	return ((unsigned int)(p1-p2));
}
// elementarni uprava bez deleni
__device__ unsigned int cuda_elem_uprava_bez_deleni2(unsigned int modul, unsigned int a_xy, unsigned int a_pp, unsigned int a_xp, unsigned int a_py)
// \STATE $a_{xy} := a_{xy} \cdot a_pp - a_xp \cdot a_py$
{
	double p1 = __fma_rd((double)a_xp, -((double)a_py), __dmul_rn( (double)a_xy, (double)a_pp ));
	double q = __double2uint_rd( p1/(double)modul );
	double p2 = __dmul_rn(q, (double)modul);
	
	return ((unsigned int)(p1-p2));
}
// elementarni uprava s delenim
__device__ unsigned int cuda_elem_uprava_s_delenim2(unsigned int modul, unsigned int a_xy, unsigned int a_xp, unsigned int a_py)
// \STATE $a_{xy} := a_xy - a_xp \cdot a_py$
{
	double p1 = __fma_rd((double)a_xp, -((double)a_py), (double)a_xy);
	double q = __double2uint_rd( p1/(double)modul );
	double p2 = __dmul_rn(q, (double)modul);
	
	return ((unsigned int)(p1-p2));
}
/* nacte/ulozi podmatici z globalni p. do sdilene nebo zpet
 * Sx, Sy - velikost podmatice, mela by se vejit do sdilene pameti
 * sx, sy - souradnice zvolene podmatice v matici, sx \in [0; ceil(N/Sx)]
 * mat_A, mat_B - zdrojova nebo cilova adresa
 */
//#define COPY_MAT_B_GLOB_TO_A_SH	1
//#define COPY_MAT_A_SH_TO_B_GLOB	2
//#define COPY_MAT_A_SH_TO_B_SH 	3
__device__ void cuda_copy_podmatice(int N, int gx, int gy, int Sx, int Sy, unsigned int* mat_A, unsigned int* mat_B, unsigned int* prava_str, int copy_to)
{
	int tid=0;
	int bdim=1;
	int itid=tid;
	unsigned int a;
	
	while(itid<Sy)
	{
		int glob_y=gy+itid;
		for(int glob_x=gx;glob_x<gx+Sx;glob_x++)
		{
			int shared_x=glob_x-gx;
			int shared_y=glob_y-gy;
	
			if(glob_x<=N && glob_y<N)
			{
				if(glob_x<N)
				{
					switch(copy_to)
					{
					case COPY_MAT_A_SH_TO_B_GLOB:
						a = mat_A[cuda_get_index(shared_x, shared_y, Sx)];
						mat_B[cuda_get_index(glob_x, glob_y, N)] = a;
						break;
					case COPY_MAT_B_GLOB_TO_A_SH:
						a = mat_B[cuda_get_index(glob_x, glob_y, N)];
						mat_A[cuda_get_index(shared_x, shared_y, Sx)] = a;
						break;
					}
				}else
				{
					switch(copy_to)
					{
					case COPY_MAT_A_SH_TO_B_GLOB:
						a = mat_A[cuda_get_index(shared_x, shared_y, Sx)];
						prava_str[glob_y] = a;
						break;
					case COPY_MAT_B_GLOB_TO_A_SH:
						a = prava_str[glob_y];
						mat_A[cuda_get_index(shared_x, shared_y, Sx)] = a;
						break;
					}
				}
			}else
			{
				if(copy_to == COPY_MAT_B_GLOB_TO_A_SH)
				{
					//if( sx==sy && ix==itid )
					//mat_A[get_index(shared_x, shared_y, Sx)] = 1;
					//else
					mat_A[cuda_get_index(shared_x, shared_y, Sx)] = 0;
				}
			}
		}
		itid+=bdim;
	}
}
/* 
 * gauss-jordanova eliminace, jednovlaknova, ve while-cyklech, primo na datech ve vstupnim poli, 
 * bez deleni - nasobim oba mergujici radky, po vypoctu kazde bunky se moduluje, 
 * dva pristupy k matici: ipivot prochazi pres matici pres radky/sloupce
 * void gauss_jordan_elim_while(int Sx, int Sy, unsigned int modul, unsigned int* m_matice)
 */
__device__ void gauss_jordan_elim_while_kernel(int Sx, int Sy, unsigned int modul, unsigned int* m_matice, unsigned int zpusob)
{
	int Smin=min(Sx, Sy);
	int tid=threadIdx.x;
	int bdim=blockDim.x;
	int itid;
	for(int ipivot=0;ipivot<Smin;ipivot++)
	{
		__shared__ int novy_pivot;
		__syncthreads();
		if(tid==0)
		{
			novy_pivot=ipivot;
			// deleni nulou => nasobeni inverznim prvkem
			if(m_matice[cuda_get_index(ipivot, ipivot, Sx)]==0)
			{
				// v 'ipivot'-tem radku na diagon�le je nula => vymena s jinym radkem
				do{
					novy_pivot++;
				}while(m_matice[cuda_get_index(ipivot, novy_pivot, Sx)]==0 && novy_pivot<Smin);
			}
		}
		__syncthreads();
		// matice je singularni
		if(novy_pivot>=Smin)
		{
			// matice nema v 'ipivot'-tem sloupci nenulovy prvek => je singularni
			//cout << "singularni" << endl;
			itid=tid;
			// singularni matice => vysledky jsou nulove (nepouzitelne)
			//while(itid<=N)
			{
					
				itid+=bdim;
			}
			return;
		}
		// musim prehodit pivotni radek s jinym
		if(novy_pivot>ipivot)
		{
			// vymena radku ipivot a novy_pivot
			itid=tid;
			unsigned int pom;
			while(itid<=Sx)
			{
				pom=m_matice[cuda_get_index(itid, ipivot, Sx)];
				m_matice[cuda_get_index(itid, ipivot, Sx)]=m_matice[cuda_get_index(itid, novy_pivot, Sx)];
				m_matice[cuda_get_index(itid, novy_pivot, Sx)]=pom;
				itid+=bdim;
			}
		}

		__syncthreads();
		unsigned int a_pp;
		if( zpusob & ZPUSOB_S_DELENIM )
		{
			unsigned int a_pp_inv = cuda_compute_inverse_eukleides(m_matice[cuda_get_index(ipivot, ipivot, Sx)], modul);
			// vydelit cely ipivot-ty radek cislem a_pp
			itid=tid;
			while(itid<Sx)
			{
				unsigned long long pom = m_matice[cuda_get_index(itid, ipivot, Sx)];
				pom *= a_pp_inv;
				pom %= modul;
				m_matice[cuda_get_index(itid, ipivot, Sx)]=(unsigned int)pom;

				itid+=bdim;
			}
		}else
		{
			a_pp = m_matice[cuda_get_index(ipivot, ipivot, Sx)];
		}

		//*
		if(zpusob & ZPUSOB_WF)
		{
			itid=tid;
			while(itid<Sy)	// prochazi jednotlive radky
			{
				if(itid!=ipivot)
				{
					unsigned int a_py = m_matice[cuda_get_index(ipivot, itid, Sx)];

					for(int iX=0;iX<Sx;iX++)	// prochazi cisla v i1-tem radku
					{
						unsigned int a_xy = m_matice[cuda_get_index(iX, itid, Sx)];
						unsigned int a_xp = m_matice[cuda_get_index(iX, ipivot, Sx)];
						if( zpusob & ZPUSOB_S_DELENIM )
						{
							m_matice[cuda_get_index(iX, itid, Sx)] = cuda_elem_uprava_s_delenim(modul, a_xy, a_xp, a_py);
						}else
						{
							m_matice[cuda_get_index(iX, itid, Sx)] = cuda_elem_uprava_bez_deleni(modul, a_xy, a_pp, a_xp, a_py);
						}
					}
				}
				itid+=bdim;
			}
		}else
		{
			for(int iY=0;iY<Sy;iY++)	// prochazi jednotlive radky
			{
				if(iY!=ipivot)
				{
					unsigned int a_py = m_matice[cuda_get_index(ipivot, iY, Sx)];
					// DEBUG
					itid=tid;
					while(itid<Sx)	// prochazi cisla v i1-tem radku
					{
						unsigned int a_xy = m_matice[cuda_get_index(itid, iY, Sx)];
						unsigned int a_xp = m_matice[cuda_get_index(itid, ipivot, Sx)];
						if( zpusob & ZPUSOB_S_DELENIM )
						{
							m_matice[cuda_get_index(itid, iY, Sx)] = cuda_elem_uprava_s_delenim(modul, a_xy, a_xp, a_py);
						}else
						{
							m_matice[cuda_get_index(itid, iY, Sx)] = cuda_elem_uprava_bez_deleni(modul, a_xy, a_pp, a_xp, a_py);
						}
						itid+=bdim;
					}
				}
				__syncthreads();
			}
		}
	}
	if( zpusob & ZPUSOB_S_DELENIM )
	{
		unsigned long long pom;
		itid=tid;
		while(itid<Smin)
		{
			pom = m_matice[cuda_get_index(Sx-1, itid, Sx)];
			pom *= cuda_compute_inverse_eukleides(m_matice[cuda_get_index(itid, itid, Sx)], modul);
			pom %= modul;
			m_matice[cuda_get_index(Sx-1, itid, Sx)] = (unsigned int)pom;
			itid+=bdim;
		}
	}
}

__global__ void kernel(int N, int* pole, int cislo)
{
	int tid=threadIdx.x;
	while(tid<N)
	{
		pole[tid]=5;
		tid+=blockDim.x;
	}
}
__global__ void cuda_GJE_while_kernel(int N, unsigned int modul, unsigned int* g_matice, unsigned int* g_prava_strana, unsigned int zpusob)
{
	int Sx=N+1;
	int Sy=N;
	extern __shared__ unsigned int shared_memory[];
	unsigned int* s_mat=&(shared_memory[0]);
	cuda_copy_podmatice(N, 0, 0, Sx, Sy, s_mat, g_matice, g_prava_strana, COPY_MAT_B_GLOB_TO_A_SH);
	gauss_jordan_elim_while_kernel(Sx, Sy, modul, s_mat, zpusob);
	cuda_copy_podmatice(N, 0, 0, Sx, Sy, s_mat, g_matice, g_prava_strana, COPY_MAT_A_SH_TO_B_GLOB);
}

__device__ void cuda_compute_podmatice24(int N, unsigned int modul, int pivot_x, int Sx, int Sy, unsigned int* s_mat, unsigned int* actions, unsigned int zpusob)
{
	// podmatice s_mat: |-- podmatice, kterou pocitam (Sx*Sy cisel) --|
	// podmatice p_mat: |-- podmatice, kterou potrebuji (az Sx^2 cisel) --|
	int minS=min(Sx,Sy);
	unsigned int* p_mat=&(s_mat[Sx*Sy]);
	unsigned int* actions1=actions;				// indexy pivotnich radku, permutace radku, 'minS' cisel
	unsigned int* actions2=&(actions1[minS]);	// cim vynasobit nebo vydelit pivotni radek; 'minS' cisel
	unsigned int* actions3=&(actions2[minS]);	// multiplikatory, 'Smin*Sy' cisel
	//cout << "modul = " << modul << endl;
	int tid=threadIdx.x;
	int bdim=blockDim.x;
	// p_mat - pomocna podmatice, max velikost Sx*Sy
	
	for(int isloupec=0;(isloupec<minS);isloupec++)
	{
		unsigned int* pom_mat;
		bool is_podm3;
		__syncthreads();
		if( (zpusob & PODMATICE_12) )
		{
			// podmatice2
			pom_mat=s_mat;
			is_podm3 = false;
			// deleni: radek sdiag na '1'
			if( zpusob & ZPUSOB_S_DELENIM )
			{
				unsigned int a_pp_inv=actions2[isloupec];
				int x=tid;
				while(x<Sx)
				{
					unsigned long long pom = s_mat[cuda_get_index(x, isloupec, Sx)];
					pom *= a_pp_inv;
					pom %= modul;
					s_mat[cuda_get_index(x, isloupec, Sx)]=(unsigned int)pom;
					x+=bdim;
				}
			}
		}else
		{
			// podmatice4
			pom_mat=p_mat;
			is_podm3 = true;
		}
		// -------------------
		unsigned int a_pp = actions2[isloupec];
		__syncthreads();
		int iY=tid;
		while(iY<Sy)
		{
			if( is_podm3 || iY!=isloupec )	// neupravuji pivotni radek pokud je podmatice1
			{
				unsigned int a_py = actions3[isloupec*Sy+iY];
				for(int iX=0;iX<Sx;iX++)
				{
					unsigned int a_xy = s_mat[cuda_get_index(iX, iY, Sx)];
					unsigned int a_xp = pom_mat[cuda_get_index(iX, isloupec, Sx)];
					//cout << "  " << a_xy << " * " << a_pp << " - " << a_xp << " * " << a_py << endl;
					if(zpusob & ZPUSOB_S_DELENIM)
					{
						s_mat[cuda_get_index(iX, iY, Sx)] = cuda_elem_uprava_s_delenim(modul, a_xy, a_xp, a_py);
					}else
					{
						s_mat[cuda_get_index(iX, iY, Sx)] = cuda_elem_uprava_bez_deleni(modul, a_xy, a_pp, a_xp, a_py);
					}
				}
			}else
			{
			}
			iY+=bdim;
		}
	}
	
}

__device__ void cuda_compute_podmatice13(int N, unsigned int modul, int pivot_x, int Sx, int Sy, unsigned int* s_mat, unsigned int* actions, unsigned int zpusob)
{
	// podmatice s_mat: |-- podmatice, kterou pocitam (Sx*Sy cisel) --|
	// podmatice p_mat: |-- podmatice, kterou potrebuji (az Sx^2 cisel) --|
	int minS=min(Sx,Sy);
	unsigned int* p_mat=&(s_mat[Sx*Sy]);
	unsigned int* actions1=&(actions[0]);				// indexy pivotnich radku, permutace radku, 'minS' cisel
	unsigned int* actions2=&(actions1[minS]);	// cim vynasobit nebo vydelit pivotni radek; 'minS' cisel
	unsigned int* actions3=&(actions2[minS]);	// multiplikatory, 'Smin*Sy' cisel

	int tid=threadIdx.x;
	int bdim=blockDim.x;
	// p_mat - pomocn� podmatice, max velikost Sx*Sy
	
// \FOR{$p$ := $1$ do $Sx$}
	for(int isloupec=0;(isloupec<minS);isloupec++)
	{
		// najit g_diagonalu ve sloupci 'isloupec'
		
		unsigned int* pom_mat;
		bool is_podm3;
		// TODO: permutace radku, aby na diagonale nebyla nula
		actions1[isloupec]=isloupec;
	
		__syncthreads();
		if( zpusob & PODMATICE_12 )	// diagonalni prvek je v 'isloupec'-tem sloupci v aktualni podmatici
		{
			// podmatice1
			pom_mat=s_mat;
			is_podm3 = false;
			// deleni: radek sdiag na '1'
			if( (tid==0) && (zpusob & ZPUSOB_S_DELENIM) )
			{
				actions2[isloupec]=cuda_compute_inverse_eukleides(s_mat[cuda_get_index(isloupec, isloupec, Sx)], modul);
			}
			__syncthreads();
			if( zpusob & ZPUSOB_S_DELENIM )
			{
				unsigned int a_pp_inv=actions2[isloupec];
				int x=tid;
				while(x<Sx)
				{
					unsigned long long pom = s_mat[cuda_get_index(x, isloupec, Sx)];
					pom *= a_pp_inv;
					pom %= modul;
					s_mat[cuda_get_index(x, isloupec, Sx)]=(unsigned int)pom;
					x+=bdim;
				}
				
			}
		}else	// 'isloupec'-ty sloupec v podmatici je pod nebo nad diagonalnim prvkem
		{
			// podmatice3
			pom_mat=p_mat;
			is_podm3 = true;
		}
		

		//vypsat_mat<unsigned int>(Sx, Sy, s_mat, NULL);
		// -------------------
		if( tid==0 && !(zpusob & ZPUSOB_S_DELENIM) )
		{
			actions2[isloupec] = pom_mat[cuda_get_index(isloupec, isloupec, Sx)];
		}
		unsigned int a_pp=1;
		if( !(zpusob & ZPUSOB_S_DELENIM) )
		{
			__syncthreads();
			a_pp=actions2[isloupec];
		}
		__syncthreads();
		int iY=tid;
		while(iY<Sy)
		{
			unsigned int a_py;
			if( is_podm3 || iY!=isloupec )	// neupravuji pivotni radek pokud je podmatice1
			{
				a_py = s_mat[cuda_get_index(isloupec, iY, Sx)];
				// TODO: ulozit a_pp, a_py
				actions3[isloupec*Sy+iY]=a_py;
				//cout << "SAVE(" << a_pp << ", " << a_py << ")" << endl;
				if(a_py!=0)	// TODO: tuhle podminku dat do podm24
				{
					for(int iX=0;iX<Sx;iX++)
					{
						unsigned int a_xy = s_mat[cuda_get_index(iX, iY, Sx)];
						unsigned int a_xp = pom_mat[cuda_get_index(iX, isloupec, Sx)];
						//cout << "  " << a_xy << " * " << a_pp << " - " << a_xp << " * " << a_py << endl;
						if(a_xp!=0)
						{
							if(zpusob & ZPUSOB_S_DELENIM)
							{
								s_mat[cuda_get_index(iX, iY, Sx)] = cuda_elem_uprava_s_delenim(modul, a_xy, a_xp, a_py);
							}else
							{
								s_mat[cuda_get_index(iX, iY, Sx)] = cuda_elem_uprava_bez_deleni(modul, a_xy, a_pp, a_xp, a_py);
							}
						}
					}
				}
			}else
			{
				actions3[isloupec*Sy+iY]=0;
			}
			iY+=bdim;
		}
		
	}
	//*/
	
}

__global__ void cuda_GJE_podmatice_kernel(int N, int Sx, int Sy, unsigned int modul, unsigned int* m_matice, unsigned int* m_prava_strana, unsigned int zpusob)
{
	extern __shared__ unsigned int shared_memory[];
	//int bid = blockIdx.x;
	//int gdim = gridDim.x;
	int Smin=min(Sx, Sy);
	unsigned int* s_matice=&(shared_memory[0]);	// velikost Sx*Sy+Sx*Smin = Sx*(Sy+Smin)
	unsigned int* actions=&(shared_memory[Sx*(Sy+Smin)]);	// velikost Smin*Sy+2*Smin
	unsigned int mask_copy = 0x0000;
	if( !(zpusob & ZPUSOB_S_DELENIM) ) mask_copy |= COPY_MAT_BEZ_DELENI;
	unsigned int* p_matice=&(s_matice[Sx*Sy]);
	
// \FOR{$p$ := $1$ do $\lceil\frac{N}{\min(S_x, S_y)}\rceil$}
	for(int ipivot=0;ipivot<N;ipivot+=Smin)
	{
	// \STATE \COMMENT{zpracovani radku, kde je Z=1}
	// \STATE nacist a spocitat $podmatice_{pp}$ \COMMENT{Z=1}
		int Py=ipivot;

		cuda_copy_podmatice(N, ipivot, Py, Sx, Sy, s_matice, m_matice, m_prava_strana, COPY_MAT_B_GLOB_TO_A_SH);
		//__syncthreads();
		// todo: compute_podmatice1
		cuda_compute_podmatice13(N, modul, ipivot, Sx, Sy, s_matice, actions, zpusob | PODMATICE_12);
		//__syncthreads();
		cuda_copy_podmatice(N, ipivot, Py, Sx, Sy, s_matice, m_matice, m_prava_strana, COPY_MAT_A_SH_TO_B_GLOB);
		//__syncthreads();
	// \FOR{$x$ := $p+1$ do $\lceil\frac{N+1}{S_x}\rceil$}
		for(int x=ipivot+Sx;x<N+1;x+=Sx)
		{
		// \STATE nacist a aplikovat operace v $actions$ na $podmatice_{xp}$ \COMMENT{Z=2}
			cuda_copy_podmatice(N, x, Py, Sx, Sy, s_matice, m_matice, m_prava_strana, COPY_MAT_B_GLOB_TO_A_SH);
			//__syncthreads();
			// todo: compute_podmatice2
			cuda_compute_podmatice24(N, modul, x, Sx, Sy, s_matice, actions, zpusob | PODMATICE_12);
			//__syncthreads();
			cuda_copy_podmatice(N, x, Py, Sx, Sy, s_matice, m_matice, m_prava_strana, COPY_MAT_A_SH_TO_B_GLOB);
			//__syncthreads();
		}
	//\ENDFOR
	// \STATE \COMMENT{zpracovani ostatnich radku}
	// \FOR{$y$ := $1$ do $\lceil\frac{N}{S_y}\rceil$}
		for(int y=0;y<N;y+=Sy)
		{
		// \IF{$y$ != $p$}
			//if(y!=Py)
			{
				// TODO: nacitani p_matice: 1) nacitat v jedne funkci spolu s s_matice (problem: rozlisovani kdy nacitat a kdy ne)
				//                          2) nacitat ve fci copy_podmatice (nebo nejake spec. fci), problem s umistenim ve velke matici
				// int Py=max(0, min(Sx, Sy*y-Sx*ipivot));
				int Py1, Sy1=0;
				int Py2, Sy2=0;
				if(y+Sy<=ipivot)
				{
					// cela podmatice je nad diagonalou ve velke matici
					Sy1=Sy;
					Py1=y;
				}else if(y<ipivot)
				{
					// cast bude nad a cast pod diagonalou
					Sy1=ipivot-y;
					Py1=y;
					Sy2=Sy-Sy1;
					Py2=y+Sy1+Sy;
				}else	// y>=ipivot
				{
					// cela podmatice bude pod diagonalou
					Sy1=Sy;
					Py1=y+Sy;
				}
				if(Py1>=N) break;
				__syncthreads();
			// \STATE nacist a vynulovat $podmatice_{py}$; \COMMENT{Z=3}
				cuda_copy_podmatice(N, ipivot, Py1, Sx, Sy1, s_matice, m_matice, m_prava_strana, COPY_MAT_B_GLOB_TO_A_SH);
				if(Sy2>0) cuda_copy_podmatice(N, ipivot, Py2, Sx, Sy2, &(s_matice[Sx*Sy1]), m_matice, m_prava_strana, COPY_MAT_B_GLOB_TO_A_SH);
				//__syncthreads();
				// todo: nenacitat prvky, ktere uz jsou v s_matice
				cuda_copy_podmatice(N, ipivot, Py, Sx, Smin, p_matice, m_matice, m_prava_strana, COPY_MAT_B_GLOB_TO_A_SH );
				//__syncthreads();
				// todo: compute_podmatice3
				cuda_compute_podmatice13(N, modul, ipivot, Sx, Sy, s_matice, actions, zpusob);
				//__syncthreads();

				cuda_copy_podmatice(N, ipivot, Py1, Sx, Sy1, s_matice, m_matice, m_prava_strana, COPY_MAT_A_SH_TO_B_GLOB);
				if(Sy2>0) cuda_copy_podmatice(N, ipivot, Py2, Sx, Sy2, &(s_matice[Sx*Sy1]), m_matice, m_prava_strana, COPY_MAT_A_SH_TO_B_GLOB);
				//__syncthreads();
			// \FOR{$x$ := $p+1$ do $\lceil\frac{N+1}{S_x}\rceil$}
				for(int x=ipivot+Sx;x<N+1;x+=Sx)
				{
				// \STATE nacist a aplikovat operace v $actions$ na $podmatice_{xy}$; \COMMENT{Z=4}
					cuda_copy_podmatice(N, x, Py1, Sx, Sy1, s_matice, m_matice, m_prava_strana, COPY_MAT_B_GLOB_TO_A_SH);
					if(Sy2>0) cuda_copy_podmatice(N, x, Py2, Sx, Sy2, &(s_matice[Sx*Sy1]), m_matice, m_prava_strana, COPY_MAT_B_GLOB_TO_A_SH);
					cuda_copy_podmatice(N, x, Py, Sx, Smin, p_matice, m_matice, m_prava_strana, COPY_MAT_B_GLOB_TO_A_SH );
					//__syncthreads();
					// todo: compute_podmatice4
					cuda_compute_podmatice24(N, modul, x, Sx, Sy, s_matice, actions, zpusob);
					//__syncthreads();
					cuda_copy_podmatice(N, x, Py1, Sx, Sy1, s_matice, m_matice, m_prava_strana, COPY_MAT_A_SH_TO_B_GLOB);
					if(Sy2>0) cuda_copy_podmatice(N, x, Py2, Sx, Sy2, &(s_matice[Sx*Sy1]), m_matice, m_prava_strana, COPY_MAT_A_SH_TO_B_GLOB);
					//__syncthreads();
				}
			// \ENDFOR
			}
		// \ENDIF
		}
	//\ENDFOR
	}
}

void cuda_GJE_podmatice(int N, unsigned int modul, unsigned int* m_matice, unsigned int* m_prava_strana, unsigned int zpusob)
{
	if(num_of_gpu<=0) return;
	int T=(gpu_property.sharedMemPerBlock / sizeof(unsigned int));
	unsigned int *g_matice, *g_prava_strana;
	hipProfilerStart();
	hipMalloc((void**)&g_matice, (N*N)*sizeof(unsigned int));
	hipMalloc((void**)&g_prava_strana, N*sizeof(unsigned int));
	hipMemcpy(g_matice, m_matice, (N*N)*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(g_prava_strana, m_prava_strana, N*sizeof(unsigned int), hipMemcpyHostToDevice);
	cuda_start_measuring();
	int num_of_threads;
	switch( ((zpusob & ZPUSOB_VLAKNA) >> 2) )
	{
	case 0:
		num_of_threads=1;
		break;
	case 1:
		num_of_threads=32;
		break;
	case 2:
		num_of_threads=128;
		break;
	case 3:
		num_of_threads = min( gpu_property.warpSize*((int)ceil((float)(N+1)/gpu_property.warpSize)), gpu_property.maxThreadsPerBlock );
		break;
	}
	
	int Nt=(int)floor((sqrt(1.0+4*(double)T)-1.0)/2.0);
	if(N<=Nt)
	{
		cuda_GJE_while_kernel<<<1,num_of_threads,(N*(N+1))*sizeof(unsigned int)>>>(N, modul, g_matice, g_prava_strana, zpusob);
	}else
	{
		int Sx;
		int Sy;
		float fSx=sqrt( (float)((N+1)*(N+1)+T) ) - (N+1);
		if( fSx < 1)
		{
			Sx=1;
			Sy=(T-3)/2;
		}else
		{
		Sx=(int)floor(fSx);
			Sy=N;
		}
		int Smin=min(Sx,Sy);
		int size_of_shared= Sx*Sy+Sx*Smin +	// size_s_matice
							Smin*Sy+2*Smin;	// size_actions
		// 
		cuda_GJE_podmatice_kernel<<<1,num_of_threads,size_of_shared*sizeof(unsigned int)>>>(N, Sx, Sy, modul, g_matice, g_prava_strana, zpusob);
	}
	hipDeviceSynchronize();
	cuda_stop_measuring();
	
	hipMemcpy(m_matice, g_matice, (N*N)*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(m_prava_strana, g_prava_strana, N*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipFree(g_matice);
	hipFree(g_prava_strana);

	hipProfilerStop();
}
__global__ void find_inverse(int N, unsigned int modul, int posl_ipivot, unsigned int* m_matice, int* pivot_radek, unsigned int* inverse)
{
	int q=posl_ipivot;
	int novy_ipivot=posl_ipivot;
	novy_ipivot++;
	unsigned int a;
	do
	{
		q++;
		a = m_matice[cuda_get_index(novy_ipivot, q, N)];
	}while( a==0 );
	unsigned int a_inv = cuda_compute_inverse_eukleides(a, modul);

	pivot_radek[0] = q;
	inverse[0] = a_inv;
}
__global__ void cuda_GJE_radky_kernel(int N, unsigned int modul, int ipivot, unsigned int* m_matice, unsigned int* m_prava_strana, 
						int* pivot_radek, unsigned int* inverse, unsigned int zpusob)
{
	int tid=threadIdx.x;
	int bdim=blockDim.x;
	int bid=blockIdx.x;
	int gdim=gridDim.x;
	int bN;
#if defined(SHARED_SIZE) && SHARED_SIZE>0
	bN=SHARED_SIZE;
#else
	bN=(int)ceil((double)(N+1)/gdim);
#endif
	//if(bid==0) return;
	// TODO: velikost sh_mem udelat konstantni - zmerit rychlost vypoctu v zavislosti na teto velikosti
#if defined(SHARED_SIZE) && SHARED_SIZE>0
	__shared__ unsigned int sh_mem[SHARED_SIZE];	// size_sh_mem = N
#endif
	__shared__ int sh_q;	// CUDA: 'q' sdilene, pak si musi kazde vlakno vzit svou kopii
	__shared__ unsigned int sh_a_pq_inv;
	
// \STATE \COMMENT{priprava pivotniho radku}
// \STATE nacist prvek $[p;q]$ do sdilene pameti
	if( tid==0 )
	{
		sh_q = pivot_radek[0];
		sh_a_pq_inv = inverse[0];
	}
	__syncthreads();
	int q = pivot_radek[0];
	unsigned int a_pq_inv=inverse[0];

// \FOR{$x$ := $p+1$ do $N$}
	int iX=tid;
	while(iX<bN)
	{
		int gX=iX+bid*bN;
		if( gX>=ipivot && gX<=N )
		{
		// \STATE nacist, vydelit a ulozit do sdilene pameti
			unsigned long long a;
			if(gX==N) a = m_prava_strana[q];
			else a = m_matice[cuda_get_index(gX, q, N)];
			a = cuda_multiply_add_modulo(modul, a, a_pq_inv, 0);
#if defined(SHARED_SIZE) && SHARED_SIZE>0
			sh_mem[iX] = (unsigned int)a;
#else
			if(gX==N) m_prava_strana[q] = (unsigned int)a;
			else m_matice[cuda_get_index(gX, q, N)] = (unsigned int)a;
#endif
		}
		iX+=bdim;
	}
	__syncthreads();
// \ENDFOR
// \FOR{$y$ := $1$ do $N$}
		// TODO: prehodit cykly pres Y a X mezi sebou, vlakno nebude prochazet radek, ale sloupec
		int iY=tid;	// prochazi pres Y, kazde vlakno samostatny radek
		while(iY<N)
		{
			unsigned int a_py = m_matice[cuda_get_index(ipivot, iY, N)];
		// \FOR{$x$ := $p+1$ do $N$}
			for(int iX=0;iX<bN;iX++)
			{
				int gX=iX+bid*bN;
				if( gX>ipivot && gX<=N )
				{
				// \IF{$y$ == $q$}
					if(iY == q)	// ma na starosti pivotni radek => pouze uklada do globalni
					{
					// \STATE ulozit do globalni pameti prvek $[x;y]=[x;q]$
#if defined(SHARED_SIZE) && SHARED_SIZE>0
						if(gX==N) m_prava_strana[iY] = sh_mem[iX];
						else m_matice[cuda_get_index(gX, iY, N)] = sh_mem[iX];
#endif
				// \ELSE
					}else
					{
					// \STATE upravit prvek $[x;y]$ stejne jako pri nulovani prvku $[p;y]$
						unsigned int a_xy;
						if(gX==N) a_xy = m_prava_strana[iY];
						else a_xy = m_matice[cuda_get_index(gX, iY, N)];
						unsigned int a_xp;
#if defined(SHARED_SIZE) && SHARED_SIZE>0
						a_xp = sh_mem[iX];
#else
						if(gX==N) a_xp=m_prava_strana[q];
						else a_xp=m_matice[cuda_get_index(gX, q, N)];
#endif
						//cout << "  " << a_xy << " * " << a_pp << " - " << a_xp << " * " << a_py << endl;
						
						a_xy = cuda_elem_uprava_s_delenim(modul, a_xy, a_xp, a_py);
						
						if(gX==N) m_prava_strana[iY] = a_xy;
						else m_matice[cuda_get_index(gX, iY, N)] = a_xy;
					}
				}
			// \ENDIF
		// \ENDFOR
			}
			iY+=bdim;
		}
// \ENDFOR

}
	
void cuda_GJE_radky(int N, unsigned int modul, unsigned int* m_matice, unsigned int* m_prava_strana, unsigned int zpusob)
{
	if(num_of_gpu<=0) return;
	int T=(gpu_property.sharedMemPerBlock / sizeof(unsigned int));
	unsigned int *g_matice, *g_prava_strana;
	hipProfilerStart();
	hipMalloc((void**)&g_matice, (N*N)*sizeof(unsigned int));
	hipMalloc((void**)&g_prava_strana, N*sizeof(unsigned int));
	hipMemcpy(g_matice, m_matice, (N*N)*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(g_prava_strana, m_prava_strana, N*sizeof(unsigned int), hipMemcpyHostToDevice);
	unsigned int* g_inverse;
	unsigned int* m_inverse;
	hipMalloc((void**)&g_inverse, sizeof(unsigned int));
	int* g_pivot;
	int* m_pivot;
	hipMalloc((void**)&g_pivot, sizeof(int));
	cuda_start_measuring();
	int num_of_blocks;
#if defined(SHARED_SIZE) && SHARED_SIZE>0
	num_of_blocks = (int)ceil((double)(N+1)/SHARED_SIZE);
#else
	num_of_blocks = gpu_property.multiProcessorCount;
#endif
	// N+1 vlaken = N radku + 1 vl na pocitani inverze
	int num_of_threads = min( gpu_property.warpSize*((int)ceil((float)(N+1)/((float)gpu_property.warpSize))), gpu_property.maxThreadsPerBlock );

	//cuda_GJE_radky_kernel<<<1,num_of_threads,size_of_shared*sizeof(unsigned int)>>>(N, modul, g_matice, g_prava_strana, zpusob);

	//find_inverse(N, modul, -1, g_matice, g_pivot, g_inverse);
// \FOR{$p$ := $1$ do $N$}
#ifdef _DEBUG
	FILE* file=fopen("log", "w");
#endif
	for(int ipivot=0;ipivot<N;ipivot++)
	{
		//printf("pivot = %d\n", ipivot);
	// \STATE \COMMENT{nalezeni radku s nenulovou hodnotou prvku $[p;q]$, kde $p<=q$}
		find_inverse<<<1,1>>>(N, modul, ipivot-1, g_matice, g_pivot, g_inverse);
		hipDeviceSynchronize();
	// \STATE \COMMENT{priprava pivotniho radku, Uprava ostatnich radku}
		cuda_GJE_radky_kernel<<<num_of_blocks,num_of_threads>>>(N, modul, ipivot, g_matice, g_prava_strana, g_pivot, g_inverse, zpusob);
		hipDeviceSynchronize();
#ifdef _DEBUG
		hipMemcpy(m_matice, g_matice, (N*N)*sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipMemcpy(m_prava_strana, g_prava_strana, N*sizeof(unsigned int), hipMemcpyDeviceToHost);
		vypsat_mat(N, N, m_matice, m_prava_strana);
		fprintf(file, "pivot=%d\n", ipivot);
		save_matrix(N, m_matice, m_prava_strana, file);
	}
	fclose(file);
#else
	}
#endif
// \ENDFOR
	
	cuda_stop_measuring();
	
	hipMemcpy(m_matice, g_matice, (N*N)*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(m_prava_strana, g_prava_strana, N*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipFree(g_matice);
	hipFree(g_prava_strana);

	hipProfilerStop();
}

void init_gpu_compute(void)
{
	extern hipEvent_t cuda_start;
	extern hipEvent_t cuda_stop;
	num_of_gpu=0;
    hipGetDeviceCount( &num_of_gpu);
	if (0<num_of_gpu) hipGetDeviceProperties( &gpu_property, 0);
	hipEventCreate(&cuda_start);
	hipEventCreate(&cuda_stop);
	//cudaProfilerInitialize
}
void print_gpus_info(void)
{
	hipDeviceProp_t prop;
    int count=0;
 
    hipGetDeviceCount( &count);
	printf("Pocet CUDA zarizeni: %d\n", count);
	for (int i=0; i< count; i++)
	{
		hipGetDeviceProperties( &prop, i);

		printf( " --- General Information for device %d ---\n", i );
		printf( "Name: %s\n", prop.name );
		printf( "Compute capability: %d.%d\n", prop.major, prop.minor );
		printf( "Clock rate: %d\n", prop.clockRate );
		printf( "Device copy overlap: " );
       
		if (prop.deviceOverlap)
			printf( "Enabled\n" );
		else
			printf( "Disabled\n" );
		printf( "Kernel execition timeout : " );


		if (prop.kernelExecTimeoutEnabled)
			printf( "Enabled\n" );
		else
			printf( "Disabled\n" );
    
		printf( " --- Memory Information for device %d ---\n", i );
		printf( "Total global mem: %ld\n", prop.totalGlobalMem );
		printf( "Total constant Mem: %ld\n", prop.totalConstMem );
		printf( "Max mem pitch: %ld\n", prop.memPitch );
		printf( "Texture Alignment: %ld\n", prop.textureAlignment );
		printf( " --- MP Information for device %d ---\n", i );
		printf( "Multiprocessor count: %d\n",
		prop.multiProcessorCount );
		printf( "Shared mem per mp: %ld\n", prop.sharedMemPerBlock );
		printf( "Registers per mp: %d\n", prop.regsPerBlock );
		printf( "Threads in warp: %d\n", prop.warpSize );
		printf( "Max threads per block: %d\n",
		prop.maxThreadsPerBlock );
		printf( "Max thread dimensions: (%d, %d, %d)\n",
		prop.maxThreadsDim[0], prop.maxThreadsDim[1],
		prop.maxThreadsDim[2] );
		printf( "Max grid dimensions: (%d, %d, %d)\n",
		prop.maxGridSize[0], prop.maxGridSize[1],
		prop.maxGridSize[2] );
		printf( "\n" );
	}
}

void print_cuda_err(hipError_t cudaErr)
{
	switch(cudaErr)
	{
	case hipSuccess: printf("hipSuccess");
		break;
	case hipErrorInvalidValue: printf("hipErrorInvalidValue");
		break;
	case hipErrorInvalidDevicePointer: printf("hipErrorInvalidDevicePointer");
		break;
	case hipErrorInvalidMemcpyDirection: printf("hipErrorInvalidMemcpyDirection");
		break;
	}
}

__global__ void cuda_GJE_global1(int N, unsigned int modul, unsigned int* g, unsigned int zpusob)
{
	gauss_jordan_elim_while_kernel(N+1, N, modul, g, zpusob);
}
void cuda_GJE_global(int N, unsigned int modul, unsigned int* m_matice, unsigned int zpusob)
{
	if(num_of_gpu<=0) return;
	unsigned int *g;
	
	hipProfilerStart();
	hipMalloc((void**)&g, (N*(N+1))*sizeof(unsigned int));
	hipMemcpy(g, m_matice, (N*(N+1))*sizeof(unsigned int), hipMemcpyHostToDevice);

	int num_of_threads;
	switch( ((zpusob & ZPUSOB_VLAKNA) >> 2) )
	{
	case 0:
		num_of_threads=1;
		break;
	case 1:
		num_of_threads=32;
		break;
	case 2:
		num_of_threads=128;
		break;
	case 3:
		num_of_threads = min( 32*((int)ceil((float)(N+1)/32.0)), gpu_property.maxThreadsPerBlock );
		break;
	}
	cuda_start_measuring();
	cuda_GJE_global1<<<1,num_of_threads>>>(N, modul, g, zpusob);
	
	hipDeviceSynchronize();
	cuda_stop_measuring();
	
	hipMemcpy(m_matice, g, (N*(N+1))*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipFree(g);

	hipProfilerStop();
}

__global__ void test_elem_uprava_kernel_bez(int n, unsigned int modul)
{
	int bdim=blockDim.x;
	unsigned int a1=1298161;
	unsigned int a2;
	unsigned int a3=a1;
	for(int i=0;i<n;i+=bdim)
	{
		a3=a1;
		a1=cuda_elem_uprava_bez_deleni(modul, a1, 1293001, a2, 1269239);
		a2=a3;
	}
}
__global__ void test_elem_uprava_kernel_s(int n, unsigned int modul)
{
	int bdim=blockDim.x;
	unsigned int a1=1298161;
	unsigned int a2;
	unsigned int a3=a1;
	for(int i=0;i<n;i+=bdim)
	{
		a3=a1;
		a1=cuda_elem_uprava_s_delenim(modul, a1, a2, 1269239);
		a2=a3;
	}
}
__global__ void test_elem_uprava_kernel_bez1(int n, unsigned int modul)
{
	int bdim=blockDim.x;
	unsigned int a1=1298161;
	unsigned int a2;
	unsigned int a3=a1;
	for(int i=0;i<n;i+=bdim)
	{
		a3=a1;
		a1=cuda_elem_uprava_bez_deleni1(modul, a1, 1293001, a2, 1269239);
		a2=a3;
	}
}
__global__ void test_elem_uprava_kernel_s1(int n, unsigned int modul)
{
	int bdim=blockDim.x;
	unsigned int a1=1298161;
	unsigned int a2;
	unsigned int a3=a1;
	for(int i=0;i<n;i+=bdim)
	{
		a3=a1;
		a1=cuda_elem_uprava_s_delenim1(modul, a1, a2, 1269239);
		a2=a3;
	}
}
__global__ void test_elem_uprava_kernel_bez2(int n, unsigned int modul)
{
	int bdim=blockDim.x;
	unsigned int a1=1298161;
	unsigned int a2;
	unsigned int a3=a1;
	for(int i=0;i<n;i+=bdim)
	{
		a3=a1;
		a1=cuda_elem_uprava_bez_deleni2(modul, a1, 1293001, a2, 1269239);
		a2=a3;
	}
}
__global__ void test_elem_uprava_kernel_s2(int n, unsigned int modul)
{
	int bdim=blockDim.x;
	unsigned int a1=1298161;
	unsigned int a2;
	unsigned int a3=a1;
	for(int i=0;i<n;i+=bdim)
	{
		a3=a1;
		a1=cuda_elem_uprava_s_delenim2(modul, a1, a2, 1269239);
		a2=a3;
	}
}


void test_elem_uprava(int N, unsigned int modul, unsigned int zpusob)
{
	if(num_of_gpu<=0) return;
	hipProfilerStart();

	cuda_start_measuring();
	// vypocet
	if( (zpusob & ZPUSOB_S_DELENIM) )
	{
		test_elem_uprava_kernel_s<<<1,1>>>(N, modul);
	}else
	{
		test_elem_uprava_kernel_bez<<<1,1>>>(N, modul);
	}
	hipDeviceSynchronize();
	cuda_stop_measuring();
	hipProfilerStop();
}

void test_elem_uprava1(int N, unsigned int modul, unsigned int zpusob)
{
	if(num_of_gpu<=0) return;
	hipProfilerStart();

	cuda_start_measuring();
	// vypocet
	if( (zpusob & ZPUSOB_S_DELENIM) )
	{
		test_elem_uprava_kernel_s1<<<1,1>>>(N, modul);
	}else
	{
		test_elem_uprava_kernel_bez1<<<1,1>>>(N, modul);
	}
	hipDeviceSynchronize();
	cuda_stop_measuring();
	hipProfilerStop();
}

void test_elem_uprava2(int N, unsigned int modul, unsigned int zpusob)
{
	if(num_of_gpu<=0) return;
	hipProfilerStart();

	cuda_start_measuring();
	// vypocet
	if( (zpusob & ZPUSOB_S_DELENIM) )
	{
		test_elem_uprava_kernel_s2<<<1,1>>>(N, modul);
	}else
	{
		test_elem_uprava_kernel_bez2<<<1,1>>>(N, modul);
	}
	hipDeviceSynchronize();
	cuda_stop_measuring();
	hipProfilerStop();
}
