#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdio>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "kernels.h"
#include "time_measure.h"
#include "common.h"



__device__ int cuda_get_index(int X, int Y, int N)	// SLOUPEC, RADEK
{
	return Y*N+X;
}
__device__ unsigned int cuda_compute_inverse_eukleides(unsigned int cislo, unsigned int modul)
{
	unsigned int a, b, a1, b1, q, r;
	a = cislo;
	b = modul;
	a1 = 0;
	b1 = 1;
	int plus = 1;

	while( b!=0 )
	{
		q = a / b;
		r = a % b;
		a = b;
		b = r;
		r = a1;
		a1 = b1 + r*q;
		b1 = r;
		plus=-plus;
	}
	if( a==1 )
	{
		if( 0<plus )
		{
			return (unsigned int)b1;
		}else
		{
			return (unsigned int)(modul-b1);
		}
	}
	return (unsigned int)0;
}

// elementarni uprava s delenim
__device__ unsigned int cuda_elem_uprava_s_delenim(unsigned int modul, unsigned int a_xy, unsigned int a_xp, unsigned int a_py)
// \STATE $a_{xy} := a_{xy} - a_xp \cdot a_py$
// TODO: merit rychlosti modulovani % a __umulhi
{
	unsigned long long m1;
	unsigned long long pom;

	pom = a_xy;
	m1 = a_xp;
	
	m1 *= a_py;
	if(pom >= m1)
	{
		pom -= m1;
		pom %= modul;
	}else
	{
		m1 -= pom;
		m1 %= modul;
		pom = modul-m1;
	}
	return ((unsigned int)pom);
}
// elementarni uprava bez deleni
__device__ unsigned int cuda_elem_uprava_bez_deleni(unsigned int modul, unsigned int a_xy, unsigned int a_pp, unsigned int a_xp, unsigned int a_py)
// \STATE $a_{xy} := a_{xy} \cdot a_pp - a_xp \cdot a_py$
{
	unsigned long long m1;
	unsigned long long pom;

	pom = a_xy;
	m1 = a_xp;
	
	pom *= a_pp;
	m1 *= a_py;
	if(pom >= m1)
	{
		pom -= m1;
		pom %= modul;
	}else
	{
		m1 -= pom;
		m1 %= modul;
		pom = modul-m1;
	}
	return ((unsigned int)pom);
}
/* nacte/ulozi podmatici z globalni p. do sdilene nebo zpet
 * Sx, Sy - velikost podmatice, mela by se vejit do sdilene pameti
 * sx, sy - souradnice zvolene podmatice v matici, sx \in [0; ceil(N/Sx)]
 * mat_A, mat_B - zdrojova nebo cilova adresa
 */
//#define COPY_MAT_B_GLOB_TO_A_SH	1
//#define COPY_MAT_A_SH_TO_B_GLOB	2
//#define COPY_MAT_A_SH_TO_B_SH 	3
__device__ void cuda_copy_podmatice(int N, int gx, int gy, int Sx, int Sy, unsigned int* mat_A, unsigned int* mat_B, unsigned int* prava_str, int copy_to)
{
	int tid=0;
	int bdim=1;
	int itid=tid;
	unsigned int a;
	
	while(itid<Sy)
	{
		int glob_y=gy+itid;
		for(int glob_x=gx;glob_x<gx+Sx;glob_x++)
		{
			int shared_x=glob_x-gx;
			int shared_y=glob_y-gy;
	
			if(glob_x<=N && glob_y<N)
			{
				if(glob_x<N)
				{
					switch(copy_to)
					{
					case COPY_MAT_A_SH_TO_B_GLOB:
						a = mat_A[cuda_get_index(shared_x, shared_y, Sx)];
						mat_B[cuda_get_index(glob_x, glob_y, N)] = a;
						break;
					case COPY_MAT_B_GLOB_TO_A_SH:
						a = mat_B[cuda_get_index(glob_x, glob_y, N)];
						mat_A[cuda_get_index(shared_x, shared_y, Sx)] = a;
						break;
					}
				}else
				{
					switch(copy_to)
					{
					case COPY_MAT_A_SH_TO_B_GLOB:
						a = mat_A[cuda_get_index(shared_x, shared_y, Sx)];
						prava_str[glob_y] = a;
						break;
					case COPY_MAT_B_GLOB_TO_A_SH:
						a = prava_str[glob_y];
						mat_A[cuda_get_index(shared_x, shared_y, Sx)] = a;
						break;
					}
				}
			}else
			{
				if(copy_to == COPY_MAT_B_GLOB_TO_A_SH)
				{
					//if( sx==sy && ix==itid )
					//mat_A[get_index(shared_x, shared_y, Sx)] = 1;
					//else
					mat_A[cuda_get_index(shared_x, shared_y, Sx)] = 0;
				}
			}
		}
		itid+=bdim;
	}
}
/* 
 * gauss-jordanova eliminace, jednovlaknova, ve while-cyklech, primo na datech ve vstupnim poli, 
 * bez deleni - nasobim oba mergujici radky, po vypoctu kazde bunky se moduluje, 
 * dva pristupy k matici: ipivot prochazi pres matici pres radky/sloupce
 * void gauss_jordan_elim_while(int Sx, int Sy, unsigned int modul, unsigned int* m_matice)
 */
__device__ void gauss_jordan_elim_while_kernel(int Sx, int Sy, unsigned int modul, unsigned int* m_matice, unsigned int zpusob)
{
	int Smin=min(Sx, Sy);
	int tid=threadIdx.x;
	int bdim=blockDim.x;
	int itid;
	for(int ipivot=0;ipivot<Smin;ipivot++)
	{
		__shared__ int novy_pivot;
		__syncthreads();
		if(tid==0)
		{
			novy_pivot=ipivot;
			// deleni nulou => nasobeni inverznim prvkem
			if(m_matice[cuda_get_index(ipivot, ipivot, Sx)]==0)
			{
				// v 'ipivot'-tem radku na diagon�le je nula => vymena s jinym radkem
				do{
					novy_pivot++;
				}while(m_matice[cuda_get_index(ipivot, novy_pivot, Sx)]==0 && novy_pivot<Smin);
			}
		}
		__syncthreads();
		// matice je singularni
		if(novy_pivot>=Smin)
		{
			// matice nema v 'ipivot'-tem sloupci nenulovy prvek => je singularni
			//cout << "singularni" << endl;
			itid=tid;
			// singularni matice => vysledky jsou nulove (nepouzitelne)
			//while(itid<=N)
			{
					
				itid+=bdim;
			}
			return;
		}
		// musim prehodit pivotni radek s jinym
		if(novy_pivot>ipivot)
		{
			// vymena radku ipivot a novy_pivot
			itid=tid;
			unsigned int pom;
			while(itid<=Sx)
			{
				pom=m_matice[cuda_get_index(itid, ipivot, Sx)];
				m_matice[cuda_get_index(itid, ipivot, Sx)]=m_matice[cuda_get_index(itid, novy_pivot, Sx)];
				m_matice[cuda_get_index(itid, novy_pivot, Sx)]=pom;
				itid+=bdim;
			}
		}

		__syncthreads();
		unsigned int a_pp;
		if( zpusob & ZPUSOB_S_DELENIM )
		{
			unsigned int a_pp_inv = cuda_compute_inverse_eukleides(m_matice[cuda_get_index(ipivot, ipivot, Sx)], modul);
			// vydelit cely ipivot-ty radek cislem a_pp
			itid=tid;
			while(itid<Sx)
			{
				unsigned long long pom = m_matice[cuda_get_index(itid, ipivot, Sx)];
				pom *= a_pp_inv;
				pom %= modul;
				m_matice[cuda_get_index(itid, ipivot, Sx)]=(unsigned int)pom;

				itid+=bdim;
			}
		}else
		{
			a_pp = m_matice[cuda_get_index(ipivot, ipivot, Sx)];
		}

		//*
		if(zpusob & ZPUSOB_WF)
		{
			itid=tid;
			while(itid<Sy)	// prochazi jednotlive radky
			{
				if(itid!=ipivot)
				{
					unsigned int a_py = m_matice[cuda_get_index(ipivot, itid, Sx)];

					for(int iX=0;iX<Sx;iX++)	// prochazi cisla v i1-tem radku
					{
						unsigned int a_xy = m_matice[cuda_get_index(iX, itid, Sx)];
						unsigned int a_xp = m_matice[cuda_get_index(iX, ipivot, Sx)];
						if( zpusob & ZPUSOB_S_DELENIM )
						{
							m_matice[cuda_get_index(iX, itid, Sx)] = cuda_elem_uprava_s_delenim(modul, a_xy, a_xp, a_py);
						}else
						{
							m_matice[cuda_get_index(iX, itid, Sx)] = cuda_elem_uprava_bez_deleni(modul, a_xy, a_pp, a_xp, a_py);
						}
					}
				}
				itid+=bdim;
			}
		}else
		{
			for(int iY=0;iY<Sy;iY++)	// prochazi jednotlive radky
			{
				if(iY!=ipivot)
				{
					unsigned int a_py = m_matice[cuda_get_index(ipivot, iY, Sx)];
					// DEBUG
					itid=tid;
					while(itid<Sx)	// prochazi cisla v i1-tem radku
					{
						unsigned int a_xy = m_matice[cuda_get_index(itid, iY, Sx)];
						unsigned int a_xp = m_matice[cuda_get_index(itid, ipivot, Sx)];
						if( zpusob & ZPUSOB_S_DELENIM )
						{
							m_matice[cuda_get_index(itid, iY, Sx)] = cuda_elem_uprava_s_delenim(modul, a_xy, a_xp, a_py);
						}else
						{
							m_matice[cuda_get_index(itid, iY, Sx)] = cuda_elem_uprava_bez_deleni(modul, a_xy, a_pp, a_xp, a_py);
						}
						itid+=bdim;
					}
				}
				__syncthreads();
			}
		}
	}
	if( zpusob & ZPUSOB_S_DELENIM )
	{
		unsigned long long pom;
		itid=tid;
		while(itid<Smin)
		{
			pom = m_matice[cuda_get_index(Sx-1, itid, Sx)];
			pom *= cuda_compute_inverse_eukleides(m_matice[cuda_get_index(itid, itid, Sx)], modul);
			pom %= modul;
			m_matice[cuda_get_index(Sx-1, itid, Sx)] = (unsigned int)pom;
			itid+=bdim;
		}
	}
}

__global__ void kernel(int N, int* pole, int cislo)
{
	int tid=threadIdx.x;
	while(tid<N)
	{
		pole[tid]=5;
		tid+=blockDim.x;
	}
}
__global__ void cuda_GJE_while_kernel(int N, unsigned int modul, unsigned int* g_matice, unsigned int* g_prava_strana, unsigned int zpusob)
{
	int Sx=N+1;
	int Sy=N;
	extern __shared__ unsigned int shared_memory[];
	unsigned int* s_mat=&(shared_memory[0]);
	cuda_copy_podmatice(N, 0, 0, Sx, Sy, s_mat, g_matice, g_prava_strana, COPY_MAT_B_GLOB_TO_A_SH);
	gauss_jordan_elim_while_kernel(Sx, Sy, modul, s_mat, zpusob);
	cuda_copy_podmatice(N, 0, 0, Sx, Sy, s_mat, g_matice, g_prava_strana, COPY_MAT_A_SH_TO_B_GLOB);
}

__device__ void cuda_compute_podmatice24(int N, unsigned int modul, int pivot_x, int Sx, int Sy, unsigned int* s_mat, unsigned int* actions, unsigned int zpusob)
{
	// podmatice s_mat: |-- podmatice, kterou pocitam (Sx*Sy cisel) --|
	// podmatice p_mat: |-- podmatice, kterou potrebuji (az Sx^2 cisel) --|
	int minS=min(Sx,Sy);
	unsigned int* p_mat=&(s_mat[Sx*Sy]);
	unsigned int* actions1=actions;				// indexy pivotnich radku, permutace radku, 'minS' cisel
	unsigned int* actions2=&(actions1[minS]);	// cim vynasobit nebo vydelit pivotni radek; 'minS' cisel
	unsigned int* actions3=&(actions2[minS]);	// multiplikatory, 'Smin*Sy' cisel
	//cout << "modul = " << modul << endl;
	int tid=threadIdx.x;
	int bdim=blockDim.x;
	// p_mat - pomocna podmatice, max velikost Sx*Sy
	
	for(int isloupec=0;(isloupec<minS);isloupec++)
	{
		unsigned int* pom_mat;
		bool is_podm3;
		__syncthreads();
		if( (zpusob & PODMATICE_12) )
		{
			// podmatice2
			pom_mat=s_mat;
			is_podm3 = false;
			// deleni: radek sdiag na '1'
			if( zpusob & ZPUSOB_S_DELENIM )
			{
				unsigned int a_pp_inv=actions2[isloupec];
				int x=tid;
				while(x<Sx)
				{
					unsigned long long pom = s_mat[cuda_get_index(x, isloupec, Sx)];
					pom *= a_pp_inv;
					pom %= modul;
					s_mat[cuda_get_index(x, isloupec, Sx)]=(unsigned int)pom;
					x+=bdim;
				}
			}
		}else
		{
			// podmatice4
			pom_mat=p_mat;
			is_podm3 = true;
		}
		// -------------------
		unsigned int a_pp = actions2[isloupec];
		__syncthreads();
		int iY=tid;
		while(iY<Sy)
		{
			if( is_podm3 || iY!=isloupec )	// neupravuji pivotni radek pokud je podmatice1
			{
				unsigned int a_py = actions3[isloupec*Sy+iY];
				for(int iX=0;iX<Sx;iX++)
				{
					unsigned int a_xy = s_mat[cuda_get_index(iX, iY, Sx)];
					unsigned int a_xp = pom_mat[cuda_get_index(iX, isloupec, Sx)];
					//cout << "  " << a_xy << " * " << a_pp << " - " << a_xp << " * " << a_py << endl;
					if(zpusob & ZPUSOB_S_DELENIM)
					{
						s_mat[cuda_get_index(iX, iY, Sx)] = cuda_elem_uprava_s_delenim(modul, a_xy, a_xp, a_py);
					}else
					{
						s_mat[cuda_get_index(iX, iY, Sx)] = cuda_elem_uprava_bez_deleni(modul, a_xy, a_pp, a_xp, a_py);
					}
				}
			}else
			{
			}
			iY+=bdim;
		}
	}
	
}

__device__ void cuda_compute_podmatice13(int N, unsigned int modul, int pivot_x, int Sx, int Sy, unsigned int* s_mat, unsigned int* actions, unsigned int zpusob)
{
	// podmatice s_mat: |-- podmatice, kterou pocitam (Sx*Sy cisel) --|
	// podmatice p_mat: |-- podmatice, kterou potrebuji (az Sx^2 cisel) --|
	int minS=min(Sx,Sy);
	unsigned int* p_mat=&(s_mat[Sx*Sy]);
	unsigned int* actions1=&(actions[0]);				// indexy pivotnich radku, permutace radku, 'minS' cisel
	unsigned int* actions2=&(actions1[minS]);	// cim vynasobit nebo vydelit pivotni radek; 'minS' cisel
	unsigned int* actions3=&(actions2[minS]);	// multiplikatory, 'Smin*Sy' cisel

	int tid=threadIdx.x;
	int bdim=blockDim.x;
	// p_mat - pomocn� podmatice, max velikost Sx*Sy
	
// \FOR{$p$ := $1$ do $Sx$}
	for(int isloupec=0;(isloupec<minS);isloupec++)
	{
		// najit g_diagonalu ve sloupci 'isloupec'
		
		unsigned int* pom_mat;
		bool is_podm3;
		// TODO: permutace radku, aby na diagonale nebyla nula
		actions1[isloupec]=isloupec;
	
		__syncthreads();
		if( zpusob & PODMATICE_12 )	// diagonalni prvek je v 'isloupec'-tem sloupci v aktualni podmatici
		{
			// podmatice1
			pom_mat=s_mat;
			is_podm3 = false;
			// deleni: radek sdiag na '1'
			if( (tid==0) && (zpusob & ZPUSOB_S_DELENIM) )
			{
				actions2[isloupec]=cuda_compute_inverse_eukleides(s_mat[cuda_get_index(isloupec, isloupec, Sx)], modul);
			}
			__syncthreads();
			if( zpusob & ZPUSOB_S_DELENIM )
			{
				unsigned int a_pp_inv=actions2[isloupec];
				int x=tid;
				while(x<Sx)
				{
					unsigned long long pom = s_mat[cuda_get_index(x, isloupec, Sx)];
					pom *= a_pp_inv;
					pom %= modul;
					s_mat[cuda_get_index(x, isloupec, Sx)]=(unsigned int)pom;
					x+=bdim;
				}
				
			}
		}else	// 'isloupec'-ty sloupec v podmatici je pod nebo nad diagonalnim prvkem
		{
			// podmatice3
			pom_mat=p_mat;
			is_podm3 = true;
		}
		

		//vypsat_mat<unsigned int>(Sx, Sy, s_mat, NULL);
		// -------------------
		if( tid==0 && !(zpusob & ZPUSOB_S_DELENIM) )
		{
			actions2[isloupec] = pom_mat[cuda_get_index(isloupec, isloupec, Sx)];
		}
		unsigned int a_pp=1;
		if( !(zpusob & ZPUSOB_S_DELENIM) )
		{
			__syncthreads();
			a_pp=actions2[isloupec];
		}
		__syncthreads();
		int iY=tid;
		while(iY<Sy)
		{
			unsigned int a_py;
			if( is_podm3 || iY!=isloupec )	// neupravuji pivotni radek pokud je podmatice1
			{
				a_py = s_mat[cuda_get_index(isloupec, iY, Sx)];
				// TODO: ulozit a_pp, a_py
				actions3[isloupec*Sy+iY]=a_py;
				//cout << "SAVE(" << a_pp << ", " << a_py << ")" << endl;
				if(a_py!=0)	// TODO: tuhle podminku dat do podm24
				{
					for(int iX=0;iX<Sx;iX++)
					{
						unsigned int a_xy = s_mat[cuda_get_index(iX, iY, Sx)];
						unsigned int a_xp = pom_mat[cuda_get_index(iX, isloupec, Sx)];
						//cout << "  " << a_xy << " * " << a_pp << " - " << a_xp << " * " << a_py << endl;
						if(a_xp!=0)
						{
							if(zpusob & ZPUSOB_S_DELENIM)
							{
								s_mat[cuda_get_index(iX, iY, Sx)] = cuda_elem_uprava_s_delenim(modul, a_xy, a_xp, a_py);
							}else
							{
								s_mat[cuda_get_index(iX, iY, Sx)] = cuda_elem_uprava_bez_deleni(modul, a_xy, a_pp, a_xp, a_py);
							}
						}
					}
				}
			}else
			{
				actions3[isloupec*Sy+iY]=0;
			}
			iY+=bdim;
		}
		
	}
	//*/
	
}

__global__ void cuda_GJE_podmatice_kernel(int N, int Sx, int Sy, unsigned int modul, unsigned int* m_matice, unsigned int* m_prava_strana, unsigned int zpusob)
{
	extern __shared__ unsigned int shared_memory[];
	//int bid = blockIdx.x;
	//int gdim = gridDim.x;
	int Smin=min(Sx, Sy);
	unsigned int* s_matice=&(shared_memory[0]);	// velikost Sx*Sy+Sx*Smin = Sx*(Sy+Smin)
	unsigned int* actions=&(shared_memory[Sx*(Sy+Smin)]);	// velikost Smin*Sy+2*Smin
	unsigned int mask_copy = 0x0000;
	if( !(zpusob & ZPUSOB_S_DELENIM) ) mask_copy |= COPY_MAT_BEZ_DELENI;
	unsigned int* p_matice=&(s_matice[Sx*Sy]);
	
// \FOR{$p$ := $1$ do $\lceil\frac{N}{\min(S_x, S_y)}\rceil$}
	for(int ipivot=0;ipivot<N;ipivot+=Smin)
	{
	// \STATE \COMMENT{zpracovani radku, kde je Z=1}
	// \STATE nacist a spocitat $podmatice_{pp}$ \COMMENT{Z=1}
		int Py=ipivot;

		cuda_copy_podmatice(N, ipivot, Py, Sx, Sy, s_matice, m_matice, m_prava_strana, COPY_MAT_B_GLOB_TO_A_SH);
		//__syncthreads();
		// todo: compute_podmatice1
		cuda_compute_podmatice13(N, modul, ipivot, Sx, Sy, s_matice, actions, zpusob | PODMATICE_12);
		//__syncthreads();
		cuda_copy_podmatice(N, ipivot, Py, Sx, Sy, s_matice, m_matice, m_prava_strana, COPY_MAT_A_SH_TO_B_GLOB);
		//__syncthreads();
	// \FOR{$x$ := $p+1$ do $\lceil\frac{N+1}{S_x}\rceil$}
		for(int x=ipivot+Sx;x<N+1;x+=Sx)
		{
		// \STATE nacist a aplikovat operace v $actions$ na $podmatice_{xp}$ \COMMENT{Z=2}
			cuda_copy_podmatice(N, x, Py, Sx, Sy, s_matice, m_matice, m_prava_strana, COPY_MAT_B_GLOB_TO_A_SH);
			//__syncthreads();
			// todo: compute_podmatice2
			cuda_compute_podmatice24(N, modul, x, Sx, Sy, s_matice, actions, zpusob | PODMATICE_12);
			//__syncthreads();
			cuda_copy_podmatice(N, x, Py, Sx, Sy, s_matice, m_matice, m_prava_strana, COPY_MAT_A_SH_TO_B_GLOB);
			//__syncthreads();
		}
	//\ENDFOR
	// \STATE \COMMENT{zpracovani ostatnich radku}
	// \FOR{$y$ := $1$ do $\lceil\frac{N}{S_y}\rceil$}
		for(int y=0;y<N;y+=Sy)
		{
		// \IF{$y$ != $p$}
			//if(y!=Py)
			{
				// TODO: nacitani p_matice: 1) nacitat v jedne funkci spolu s s_matice (problem: rozlisovani kdy nacitat a kdy ne)
				//                          2) nacitat ve fci copy_podmatice (nebo nejake spec. fci), problem s umistenim ve velke matici
				// int Py=max(0, min(Sx, Sy*y-Sx*ipivot));
				int Py1, Sy1=0;
				int Py2, Sy2=0;
				if(y+Sy<=ipivot)
				{
					// cela podmatice je nad diagonalou ve velke matici
					Sy1=Sy;
					Py1=y;
				}else if(y<ipivot)
				{
					// cast bude nad a cast pod diagonalou
					Sy1=ipivot-y;
					Py1=y;
					Sy2=Sy-Sy1;
					Py2=y+Sy1+Sy;
				}else	// y>=ipivot
				{
					// cela podmatice bude pod diagonalou
					Sy1=Sy;
					Py1=y+Sy;
				}
				if(Py1>=N) break;
				__syncthreads();
			// \STATE nacist a vynulovat $podmatice_{py}$; \COMMENT{Z=3}
				cuda_copy_podmatice(N, ipivot, Py1, Sx, Sy1, s_matice, m_matice, m_prava_strana, COPY_MAT_B_GLOB_TO_A_SH);
				if(Sy2>0) cuda_copy_podmatice(N, ipivot, Py2, Sx, Sy2, &(s_matice[Sx*Sy1]), m_matice, m_prava_strana, COPY_MAT_B_GLOB_TO_A_SH);
				//__syncthreads();
				// todo: nenacitat prvky, ktere uz jsou v s_matice
				cuda_copy_podmatice(N, ipivot, Py, Sx, Smin, p_matice, m_matice, m_prava_strana, COPY_MAT_B_GLOB_TO_A_SH );
				//__syncthreads();
				// todo: compute_podmatice3
				cuda_compute_podmatice13(N, modul, ipivot, Sx, Sy, s_matice, actions, zpusob);
				//__syncthreads();

				cuda_copy_podmatice(N, ipivot, Py1, Sx, Sy1, s_matice, m_matice, m_prava_strana, COPY_MAT_A_SH_TO_B_GLOB);
				if(Sy2>0) cuda_copy_podmatice(N, ipivot, Py2, Sx, Sy2, &(s_matice[Sx*Sy1]), m_matice, m_prava_strana, COPY_MAT_A_SH_TO_B_GLOB);
				//__syncthreads();
			// \FOR{$x$ := $p+1$ do $\lceil\frac{N+1}{S_x}\rceil$}
				for(int x=ipivot+Sx;x<N+1;x+=Sx)
				{
				// \STATE nacist a aplikovat operace v $actions$ na $podmatice_{xy}$; \COMMENT{Z=4}
					cuda_copy_podmatice(N, x, Py1, Sx, Sy1, s_matice, m_matice, m_prava_strana, COPY_MAT_B_GLOB_TO_A_SH);
					if(Sy2>0) cuda_copy_podmatice(N, x, Py2, Sx, Sy2, &(s_matice[Sx*Sy1]), m_matice, m_prava_strana, COPY_MAT_B_GLOB_TO_A_SH);
					cuda_copy_podmatice(N, x, Py, Sx, Smin, p_matice, m_matice, m_prava_strana, COPY_MAT_B_GLOB_TO_A_SH );
					//__syncthreads();
					// todo: compute_podmatice4
					cuda_compute_podmatice24(N, modul, x, Sx, Sy, s_matice, actions, zpusob);
					//__syncthreads();
					cuda_copy_podmatice(N, x, Py1, Sx, Sy1, s_matice, m_matice, m_prava_strana, COPY_MAT_A_SH_TO_B_GLOB);
					if(Sy2>0) cuda_copy_podmatice(N, x, Py2, Sx, Sy2, &(s_matice[Sx*Sy1]), m_matice, m_prava_strana, COPY_MAT_A_SH_TO_B_GLOB);
					//__syncthreads();
				}
			// \ENDFOR
			}
		// \ENDIF
		}
	//\ENDFOR
	}
}

void cuda_GJE_podmatice(int N, unsigned int modul, unsigned int* m_matice, unsigned int* m_prava_strana, unsigned int zpusob)
{
	if(num_of_gpu<=0) return;
	int T=(gpu_property.sharedMemPerBlock / sizeof(unsigned int));
	unsigned int *g_matice, *g_prava_strana;
	hipProfilerStart();
	hipMalloc((void**)&g_matice, (N*N)*sizeof(unsigned int));
	hipMalloc((void**)&g_prava_strana, N*sizeof(unsigned int));
	hipMemcpy(g_matice, m_matice, (N*N)*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(g_prava_strana, m_prava_strana, N*sizeof(unsigned int), hipMemcpyHostToDevice);
	cuda_start_measuring();
	int num_of_threads;
	switch( ((zpusob & ZPUSOB_VLAKNA) >> 2) )
	{
	case 0:
		num_of_threads=1;
		break;
	case 1:
		num_of_threads=32;
		break;
	case 2:
		num_of_threads=128;
		break;
	case 3:
		num_of_threads = min( gpu_property.warpSize*((int)ceil((float)(N+1)/gpu_property.warpSize)), gpu_property.maxThreadsPerBlock );
		break;
	}
	
	int Nt=(int)floor((sqrt(1.0+4*(double)T)-1.0)/2.0);
	if(N<=Nt)
	{
		cuda_GJE_while_kernel<<<1,num_of_threads,(N*(N+1))*sizeof(unsigned int)>>>(N, modul, g_matice, g_prava_strana, zpusob);
	}else
	{
		int Sx;
		int Sy;
		float fSx=sqrt( (float)((N+1)*(N+1)+T) ) - (N+1);
		if( fSx < 1)
		{
			Sx=1;
			Sy=(T-3)/2;
		}else
		{
		Sx=(int)floor(fSx);
			Sy=N;
		}
		int Smin=min(Sx,Sy);
		int size_of_shared= Sx*Sy+Sx*Smin +	// size_s_matice
							Smin*Sy+2*Smin;	// size_actions
		// 
		cuda_GJE_podmatice_kernel<<<1,num_of_threads,size_of_shared*sizeof(unsigned int)>>>(N, Sx, Sy, modul, g_matice, g_prava_strana, zpusob);
	}
	hipDeviceSynchronize();
	cuda_stop_measuring();
	
	hipMemcpy(m_matice, g_matice, (N*N)*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(m_prava_strana, g_prava_strana, N*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipFree(g_matice);
	hipFree(g_prava_strana);

	hipProfilerStop();
}

__global__ void cuda_GJE_radky_kernel(int N, unsigned int modul, unsigned int* m_matice, unsigned int* m_prava_strana, unsigned int zpusob)
{
	int tid=threadIdx.x;
	int bdim=blockDim.x;
	//int bid=blockIdx.x;
	//int gdim=gridDim.x;

	extern __shared__ unsigned int sh_mem[];	// size_sh_mem = N+1
	__shared__ int sh_q;	// CUDA: 'q' sdilene, pak si musi kazde vlakno vzit svou kopii
	__shared__ unsigned int sh_a_pq_inv;
	int q;
	if( tid==0 ) sh_q=N;
// \FOR{$p$ := $1$ do $N$}
	// TODO: pouzit atomicInc kdyz budu pracovat s vice bloky
	for(int ipivot=0;ipivot<N;ipivot++)
	{
	// \STATE \COMMENT{nalezeni radku s nenulovou hodnotou prvku $[p;q]$, kde $p<=q$}
		int i=ipivot+tid;
			while(i<N)
			{
				if(m_matice[cuda_get_index(ipivot, i, N)]!=0)
				{
					int old = atomicMin(&sh_q, i);
					break;
				}
				i+=bdim;
			}
		//*/
		if( tid==0 ) sh_q=ipivot;
		__syncthreads();
		q=sh_q;
	// \STATE \COMMENT{priprava pivotniho radku}
	// \STATE nacist prvek $[p;q]$ do sdilene pameti
		// CUDA: shared, tid==0
		if( tid==0 )
		{
			unsigned int a_pq = m_matice[cuda_get_index(ipivot, q, N)];
			sh_mem[ipivot] = a_pq;
			sh_a_pq_inv=cuda_compute_inverse_eukleides(a_pq, modul);
		}
	// \FOR{$x$ := $p+1$ do $N$}
		__syncthreads();
		i=ipivot+tid;
		unsigned int a_pq_inv=sh_a_pq_inv;
		while(i<=N)	// CUDA: pres tid
		{
		// \STATE nacist, vydelit a ulozit do sdilene pameti
			unsigned long long a;
			if(i==N) a = m_prava_strana[q];
			else a = m_matice[cuda_get_index(i, q, N)];
			if( zpusob & ZPUSOB_S_DELENIM )
			{
				a *= a_pq_inv;
				a %= modul;
			}
			sh_mem[i] = (unsigned int)a;
			i+=bdim;
		}

		__syncthreads();
	// \ENDFOR
	// \FOR{$y$ := $1$ do $N$}
			i=tid;	// prochazi pres Y, kazde vlakno samostatny radek
			while(i<N)
			{
				unsigned int a_py = m_matice[cuda_get_index(ipivot, i, N)];
				unsigned int a_pq = sh_mem[ipivot];
			// \FOR{$x$ := $p+1$ do $N$}
				for(int iX=ipivot;iX<=N;iX++)
				{
				// \IF{$y$ == $q$}
					if(i == q)	// ma na starosti pivotni radek => pouze uklada do globalni
					{
						if( zpusob & ZPUSOB_S_DELENIM )
						{
						// \STATE ulozit do globalni pameti prvek $[x;y]=[x;q]$
							if(iX==N) m_prava_strana[i] = sh_mem[iX];
							else m_matice[cuda_get_index(iX, i, N)] = sh_mem[iX];
						}
				// \ELSE
					}else
					{
					// \STATE upravit prvek $[x;y]$ stejne jako pri nulovani prvku $[p;y]$
						unsigned int a_xy;
						if(iX==N) a_xy = m_prava_strana[i];
						else a_xy = m_matice[cuda_get_index(iX, i, N)];
						unsigned int a_xp = sh_mem[iX];
						//cout << "  " << a_xy << " * " << a_pp << " - " << a_xp << " * " << a_py << endl;
						if(zpusob & ZPUSOB_S_DELENIM)
						{
							a_xy = cuda_elem_uprava_s_delenim(modul, a_xy, a_xp, a_py);
						}else
						{
							a_xy = cuda_elem_uprava_bez_deleni(modul, a_xy, a_pq, a_xp, a_py);
						}
						if(iX==N) m_prava_strana[i] = a_xy;
						else m_matice[cuda_get_index(iX, i, N)] = a_xy;
					}
				// \ENDIF
			// \ENDFOR
				}
				if( !(zpusob & ZPUSOB_S_DELENIM) && (i<ipivot) )
				{
					// uprava diagonalniho prvku kdyz upravuju bez deleni a lezi v uz spracovanym sloupci
					unsigned long long a_xy = m_matice[cuda_get_index(i, i, N)];
					a_xy *= a_pq;
					a_xy %=modul;
					//cout << "  " << a_xy << " * " << a_pp << " - " << a_xp << " * " << a_py << endl;
					m_matice[cuda_get_index(i, i, N)] = (unsigned int)a_xy;
				}
				i+=bdim;
			}
	// \ENDFOR
		__syncthreads();
	}
// \ENDFOR
}
	
void cuda_GJE_radky(int N, unsigned int modul, unsigned int* m_matice, unsigned int* m_prava_strana, unsigned int zpusob)
{
	if(num_of_gpu<=0) return;
	unsigned int *g_matice, *g_prava_strana;
	hipProfilerStart();
	hipMalloc((void**)&g_matice, (N*N)*sizeof(unsigned int));
	hipMalloc((void**)&g_prava_strana, N*sizeof(unsigned int));
	hipMemcpy(g_matice, m_matice, (N*N)*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(g_prava_strana, m_prava_strana, N*sizeof(unsigned int), hipMemcpyHostToDevice);
	cuda_start_measuring();

	// N+1 vlaken = N radku + 1 vl na pocitani inverze
	int num_of_threads = min( gpu_property.warpSize*((int)ceil((float)(N+1)/gpu_property.warpSize)), gpu_property.maxThreadsPerBlock );
	int size_of_shared=N+1;
	cuda_GJE_radky_kernel<<<1,num_of_threads,size_of_shared*sizeof(unsigned int)>>>(N, modul, g_matice, g_prava_strana, zpusob);
	
	hipDeviceSynchronize();
	cuda_stop_measuring();
	
	hipMemcpy(m_matice, g_matice, (N*N)*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(m_prava_strana, g_prava_strana, N*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipFree(g_matice);
	hipFree(g_prava_strana);

	hipProfilerStop();
}

void init_gpu_compute(void)
{
	extern hipEvent_t cuda_start;
	extern hipEvent_t cuda_stop;
	num_of_gpu=0;
    hipGetDeviceCount( &num_of_gpu);
	if (0<num_of_gpu) hipGetDeviceProperties( &gpu_property, 0);
	hipEventCreate(&cuda_start);
	hipEventCreate(&cuda_stop);
	//cudaProfilerInitialize
}
void print_gpus_info(void)
{
	hipDeviceProp_t prop;
    int count=0;
 
    hipGetDeviceCount( &count);
	printf("Pocet CUDA zarizeni: %d\n", count);
	for (int i=0; i< count; i++)
	{
		hipGetDeviceProperties( &prop, i);

		printf( " --- General Information for device %d ---\n", i );
		printf( "Name: %s\n", prop.name );
		printf( "Compute capability: %d.%d\n", prop.major, prop.minor );
		printf( "Clock rate: %d\n", prop.clockRate );
		printf( "Device copy overlap: " );
       
		if (prop.deviceOverlap)
			printf( "Enabled\n" );
		else
			printf( "Disabled\n" );
		printf( "Kernel execition timeout : " );


		if (prop.kernelExecTimeoutEnabled)
			printf( "Enabled\n" );
		else
			printf( "Disabled\n" );
    
		printf( " --- Memory Information for device %d ---\n", i );
		printf( "Total global mem: %ld\n", prop.totalGlobalMem );
		printf( "Total constant Mem: %ld\n", prop.totalConstMem );
		printf( "Max mem pitch: %ld\n", prop.memPitch );
		printf( "Texture Alignment: %ld\n", prop.textureAlignment );
		printf( " --- MP Information for device %d ---\n", i );
		printf( "Multiprocessor count: %d\n",
		prop.multiProcessorCount );
		printf( "Shared mem per mp: %ld\n", prop.sharedMemPerBlock );
		printf( "Registers per mp: %d\n", prop.regsPerBlock );
		printf( "Threads in warp: %d\n", prop.warpSize );
		printf( "Max threads per block: %d\n",
		prop.maxThreadsPerBlock );
		printf( "Max thread dimensions: (%d, %d, %d)\n",
		prop.maxThreadsDim[0], prop.maxThreadsDim[1],
		prop.maxThreadsDim[2] );
		printf( "Max grid dimensions: (%d, %d, %d)\n",
		prop.maxGridSize[0], prop.maxGridSize[1],
		prop.maxGridSize[2] );
		printf( "\n" );
	}
}

void print_cuda_err(hipError_t cudaErr)
{
	switch(cudaErr)
	{
	case hipSuccess: printf("hipSuccess");
		break;
	case hipErrorInvalidValue: printf("hipErrorInvalidValue");
		break;
	case hipErrorInvalidDevicePointer: printf("hipErrorInvalidDevicePointer");
		break;
	case hipErrorInvalidMemcpyDirection: printf("hipErrorInvalidMemcpyDirection");
		break;
	}
}

__global__ void cuda_GJE_global1(int N, unsigned int modul, unsigned int* g, unsigned int zpusob)
{
	gauss_jordan_elim_while_kernel(N+1, N, modul, g, zpusob);
}
void cuda_GJE_global(int N, unsigned int modul, unsigned int* m_matice, unsigned int zpusob)
{
	if(num_of_gpu<=0) return;
	unsigned int *g;
	
	hipProfilerStart();
	hipMalloc((void**)&g, (N*(N+1))*sizeof(unsigned int));
	hipMemcpy(g, m_matice, (N*(N+1))*sizeof(unsigned int), hipMemcpyHostToDevice);

	int num_of_threads;
	switch( ((zpusob & ZPUSOB_VLAKNA) >> 2) )
	{
	case 0:
		num_of_threads=1;
		break;
	case 1:
		num_of_threads=32;
		break;
	case 2:
		num_of_threads=128;
		break;
	case 3:
		num_of_threads = min( 256*((int)ceil((float)(N+1)/32.0)), gpu_property.maxThreadsPerBlock );
		break;
	}
	cuda_start_measuring();
	cuda_GJE_global1<<<1,num_of_threads>>>(N, modul, g, zpusob);
	
	hipDeviceSynchronize();
	cuda_stop_measuring();
	
	hipMemcpy(m_matice, g, (N*(N+1))*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipFree(g);

	hipProfilerStop();
}

__global__ void test_elem_uprava_bez_deleni(int n, unsigned int modul)
{
	int bdim=blockDim.x;
	unsigned int a1=1298161;
	unsigned int a2;
	unsigned int a3=a1;
	for(int i=0;i<n;i+=bdim)
	{
		a3=a1;
		a1=cuda_elem_uprava_bez_deleni(modul, a1, 1293001, a2, 1269239);
		a2=a3;
	}
}
__global__ void test_elem_uprava_s_delenim(int n, unsigned int modul)
{
	int bdim=blockDim.x;
	unsigned int a1=1298161;
	unsigned int a2;
	unsigned int a3=a1;
	for(int i=0;i<n;i+=bdim)
	{
		a3=a1;
		a1=cuda_elem_uprava_s_delenim(modul, a1, a2, 1269239);
		a2=a3;
	}
}
void test_elem_uprava(int N, unsigned int modul, unsigned int zpusob)
{
	if(num_of_gpu<=0) return;
	hipProfilerStart();
	int num_of_threads=1;

	cuda_start_measuring();
	// vypocet
	if( (zpusob & ZPUSOB_S_DELENIM) )
	{
		test_elem_uprava_s_delenim<<<1,num_of_threads>>>(N, modul);
	}else
	{
		test_elem_uprava_bez_deleni<<<1,num_of_threads>>>(N, modul);
	}

	hipDeviceSynchronize();
	cuda_stop_measuring();
	
	hipProfilerStop();
}

__global__ void test_inverse_kernel(int n, unsigned int modul)
{
	int bdim=blockDim.x;
	unsigned int a1=1757161;
	for(int i=0;i<n;i+=bdim)
	{
		a1 = cuda_compute_inverse_eukleides(a1, modul);
		a1++;
		
	}
}

void test_inverse(int N, unsigned int modul)
{
	if(num_of_gpu<=0) return;
	hipProfilerStart();
	int num_of_threads=1;
	
	cuda_start_measuring();
	// vypocet
	test_inverse_kernel<<<1,num_of_threads>>>(N, modul);

	hipDeviceSynchronize();
	cuda_stop_measuring();
	
	hipProfilerStop();
}
